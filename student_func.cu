#include "hip/hip_runtime.h"
#include "utils.h"

__global__
void negateImage(const uchar4* const inputImageRGBA,
                 uchar4* const outputImageRGBA,
                 int numCols) {
  const int2 matrix = make_int2( blockIdx.x * blockDim.x + threadIdx.x,
                             blockIdx.y * blockDim.y + threadIdx.y);
  const int index = matrix.y * numCols + matrix.x;

  uchar4 output = make_uchar4(255-inputImageRGBA[index].x, 255-inputImageRGBA[index].y, 255-inputImageRGBA[index].z, 255);

  outputImageRGBA[index] = output;
}

__global__
void flipX(const uchar4* const inputImageRGBA,
           uchar4* const outputImageRGBA,
           int numCols, int numRows) {
  const int2 matrix = make_int2( blockIdx.x * blockDim.x + threadIdx.x,
                             blockIdx.y * blockDim.y + threadIdx.y);
  const int index = matrix.y * numCols + matrix.x;

  const int outputIndex = (numRows * numCols - 1) - index;

  outputImageRGBA[outputIndex] = inputImageRGBA[index];
}

__global__
void flipY(const uchar4* const inputImageRGBA,
           uchar4* const outputImageRGBA,
           int numCols, int numRows) {
  const int2 matrix = make_int2( blockIdx.x * blockDim.x + threadIdx.x,
                             blockIdx.y * blockDim.y + threadIdx.y);
  const int index = matrix.y * numCols + matrix.x;

  const int2 outputMatrix = make_int2(numCols - matrix.x - 1, matrix.y);
  const int outputIndex = outputMatrix.y * numCols + outputMatrix.x;

  outputImageRGBA[outputIndex] = inputImageRGBA[index];
}

void transform(const uchar4 * const h_inputImageRGBA, uchar4 * const d_inputImageRGBA,
                        uchar4* const d_outputImageRGBA, const size_t numRows, const size_t numCols, std::string transformation)
{
  //Set reasonable block size (i.e., number of threads per block)
  const dim3 blockSize(32, 32);


  //Compute correct grid size (i.e., number of blocks per kernel launch)
  //from the image size and and block size.
  const dim3 gridSize(numCols/blockSize.x + 1, numRows/blockSize.y + 1);

  if (transformation == "negate")
    negateImage<<<gridSize, blockSize>>>(d_inputImageRGBA, d_outputImageRGBA, numCols);
  else if (transformation == "flipX")
    flipX<<<gridSize, blockSize>>>(d_inputImageRGBA, d_outputImageRGBA, numCols, numRows);
  else if (transformation == "flipY")
    flipY<<<gridSize, blockSize>>>(d_inputImageRGBA, d_outputImageRGBA, numCols, numRows);
  
  
  hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());

}
